#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define TPB 1024

__global__ void rightNeighborSum(double *d_res, double *d_V, long int n){
    long int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n - 1) return;

    d_res[idx] = d_V[idx] + d_V[idx + 1];
}

__global__ void middleNeighborSum(double *d_res, double *d_V, long int n){
    long int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0 && idx >= n - 1) return;

    d_res[idx - 1] = d_V[idx - 1] + d_V[idx + 1];
}

void rNeighborLauncher(double *res, double *V, long int n){
	double *d_res, *d_V;

	hipMalloc(&d_V, n * sizeof(double));
	hipMalloc(&d_res, (n - 1) * sizeof(double));

	hipMemcpy(d_V, V, n * sizeof(double), hipMemcpyHostToDevice);

	rightNeighborSum<<<(n + TPB - 1) / TPB, TPB>>>(d_res, d_V, n);

	hipMemcpy(res, d_res, (n - 1)*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_V);
	hipFree(d_res);
	hipDeviceReset();
}

void mNeighborLauncher(double *res, double *V, long int n){
	double *d_res, *d_V;

	hipMalloc(&d_V, n * sizeof(double));
	hipMalloc(&d_res, (n - 2) * sizeof(double));

	hipMemcpy(d_V, V, n * sizeof(double), hipMemcpyHostToDevice);

	middleNeighborSum<<<(n + TPB - 1) / TPB, TPB>>>(d_res, d_V, n);

	hipMemcpy(res, d_res, (n - 2)*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_V);
	hipFree(d_res);
	hipDeviceReset();
}